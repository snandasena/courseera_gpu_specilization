#include "hip/hip_runtime.h"

#include "stream.h"

// Increments all of the values in the input arrays
__global__ void kernelA1(float *dev_mem, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        dev_mem[i] = dev_mem[i] + 1;
    }
}

//Doubles all the values in the input arrays
__global__ void kernelB1(float *dev_mem, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        dev_mem[i] = dev_mem[i] * 2;
    }
}

// Decrements all of the values in the input arrays
__global__ void kernelA2(float *dev_mem, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        dev_mem[i] = dev_mem[i] - 1;
    }
}

//Halves all the values in the input arrays
__global__ void kernelB2(float *dev_mem, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        dev_mem[i] = dev_mem[i] / 2;
    }
}

// This will generate an array of size numElements of random integers from 0 to 255 in pageable host memory
// The host memory has to be page-locked memory or control of streams is not guaranteed
// Note that I have added an argument for the random seed, so that you can generate the same "random" values
// for multiple runs to see the result of different actions on the same set of "random" values
__host__ float *allocateHostMemory(int numElements, int seed)
{
    seed = seed != -1 ? seed : 0;
    srand(seed);
    size_t size = numElements * sizeof(float);
    float random_max = 255.0f;

    // Allocate the host pinned memory input pointer B
    float *data;
    hipHostAlloc((void**)&data, size, hipHostMallocDefault);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        // Feel free to change the max value of the random input data by replacing 255 with a smaller or larger number
        data[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/random_max));
    }

    return data;
}

__host__ float * allocateDeviceMemory(int numElements)
{
    // Allocate the device input vector a
    float *dev_mem = NULL;
    size_t size = numElements * sizeof(float);
    hipError_t err = hipMalloc(&dev_mem, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector memory (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return dev_mem;
}

//Synchronous copy of data from host to device using a default stream
__host__ void copyFromHostToDeviceSync(float *host_mem, float *dev_mem, int numElements)
{
    size_t size = numElements * sizeof(float);
    // Copy the host input vector to the device input vectors
    printf("Copy input data from the host memory to the CUDA device\n");
    hipError_t err = hipMemcpy(dev_mem, host_mem, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

//Asynchronous copy of data from host to device using a non-default stream
__host__ void copyFromHostToDeviceAsync(float *host_mem, float *dev_mem, int numElements, hipStream_t stream)
{
    size_t size = numElements * sizeof(float);
    // Copy the host input vector to the device input vectors
    printf("Copy input data from the host memory to the CUDA device\n");
    hipError_t err = hipMemcpyAsync(dev_mem, host_mem, size, hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

//Synchronous copy of data from device to host using the default stream
__host__ void copyFromDeviceToHostSync(float *dev_mem, float *host_mem, int numElements)
{
    size_t size = numElements * sizeof(float);
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipError_t err = hipMemcpy(host_mem, dev_mem, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

//Synchronous copy of data from device to host using a non-default stream
__host__ void copyFromDeviceToHostAsync(float *dev_mem, float *host_mem, int numElements, hipStream_t stream)
{
    size_t size = numElements * sizeof(float);
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipError_t err = hipMemcpyAsync(host_mem, dev_mem, size, hipMemcpyDeviceToHost, stream);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Free device global memory
__host__ void deallocateDevMemory(float *dev_mem)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipFree(dev_mem);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ std::tuple<int, int> determineThreadBlockDimensions(int num_elements)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    return {threadsPerBlock, blocksPerGrid};
}

__host__ float * runStreamsFullAsync(float *host_mem, int num_elements)
{
    // Prepare all streams such that all kernels and memory copies execute asynchronously
    hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
    hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream3,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream4,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream5,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream6,hipStreamNonBlocking);

    // Prepare device memory based on host memory
    float *dev_mem = allocateDeviceMemory(num_elements);
    copyFromHostToDeviceAsync(host_mem, dev_mem, num_elements, stream1);

    // Execute 4 kernels asynchronously on independent streams
    auto[threadsPerBlock, blocksPerGrid] = determineThreadBlockDimensions(num_elements);
    kernelA1<<<blocksPerGrid,threadsPerBlock,0, stream2>>>(dev_mem, num_elements);
    kernelB1<<<blocksPerGrid,threadsPerBlock,0, stream3>>>(dev_mem, num_elements);
    kernelA2<<<blocksPerGrid,threadsPerBlock,0, stream4>>>(dev_mem, num_elements);
    kernelB2<<<blocksPerGrid,threadsPerBlock,0, stream5>>>(dev_mem, num_elements);

    // Copy device memory back to host asynchronously
    copyFromDeviceToHostAsync(dev_mem, host_mem, num_elements, stream6);
    deallocateDevMemory(dev_mem);

    // Wait for all streams to be completed
    // This might act differently on multiple GPU system without assigning devices
    hipDeviceSynchronize();

    return host_mem;
}

__host__ float * runStreamsBlockingKernel2StreamsNaive(float *host_mem, int num_elements)
{
    // Prepare all streams such that all kernels and memory copies execute asynchronously
    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream3,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream4,hipStreamNonBlocking);

    // Prepare device memory based on host memory
    float *dev_mem = allocateDeviceMemory(num_elements);
    copyFromHostToDeviceAsync(host_mem, dev_mem, num_elements, stream1);

    // Execute 2 pairs of kernels asynchronous with resepect to their streams
    // The order of execution can have an effect on the blocking behavious
    auto[threadsPerBlock, blocksPerGrid] = determineThreadBlockDimensions(num_elements);
    kernelA1<<<blocksPerGrid,threadsPerBlock,0, stream2>>>(dev_mem, num_elements);
    kernelB1<<<blocksPerGrid,threadsPerBlock,0, stream2>>>(dev_mem, num_elements);
    kernelA2<<<blocksPerGrid,threadsPerBlock,0, stream3>>>(dev_mem, num_elements);
    kernelB2<<<blocksPerGrid,threadsPerBlock,0, stream3>>>(dev_mem, num_elements);

    // Copy device memory back to host asynchronously
    copyFromDeviceToHostAsync(dev_mem, host_mem, num_elements, stream4);
    deallocateDevMemory(dev_mem);

    // Wait for all streams to be completed
    // This might act differently on multiple GPU system without assigning devices
    hipDeviceSynchronize();

    return host_mem;
}

__host__ float * runStreamsBlockingKernel2StreamsOptimal(float *host_mem, int num_elements)
{
    // Prepare all streams such that all kernels and memory copies execute asynchronously
    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream3,hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream4,hipStreamNonBlocking);

    // Prepare device memory based on host memory
    float *dev_mem = allocateDeviceMemory(num_elements);
    copyFromHostToDeviceAsync(host_mem, dev_mem, num_elements, stream1);

    // Execute 2 pairs of kernels asynchronous with resepect to their streams
    // The order of execution can have an effect on the blocking behavious
    auto[threadsPerBlock, blocksPerGrid] = determineThreadBlockDimensions(num_elements);
    kernelA1<<<blocksPerGrid,threadsPerBlock,0, stream2>>>(dev_mem, num_elements);
    kernelA2<<<blocksPerGrid,threadsPerBlock,0, stream3>>>(dev_mem, num_elements);
    kernelB1<<<blocksPerGrid,threadsPerBlock,0, stream2>>>(dev_mem, num_elements);
    kernelB2<<<blocksPerGrid,threadsPerBlock,0, stream3>>>(dev_mem, num_elements);

    // Copy device memory back to host asynchronously
    copyFromDeviceToHostAsync(dev_mem, host_mem, num_elements, stream4);
    deallocateDevMemory(dev_mem);

    // Wait for all streams to be completed
    // This might act differently on multiple GPU system without assigning devices
    hipDeviceSynchronize();

    return host_mem;
}

__host__ void printHostMemory(float *host_mem, int num_elments)
{
    // Output results
    for(int i = 0; i < num_elments; i++)
    {
        printf("%.6f ",host_mem[i]);
    }
    printf("\n");
}

int main()
{
    int num_elements = 255; // Can be altered but keep it less than 1/2 the memory size of global memory for full concurrency
    int rand_seed = 0; // You can set this to different values for each run but default will be the same to see the effect on data

    float * host_mem = allocateHostMemory(num_elements, rand_seed);
    host_mem = runStreamsFullAsync(host_mem, num_elements);
    printHostMemory(host_mem, num_elements);

    host_mem = allocateHostMemory(num_elements, 0);
    host_mem = runStreamsBlockingKernel2StreamsNaive(host_mem, num_elements);
    printHostMemory(host_mem, num_elements);

    host_mem = allocateHostMemory(num_elements, 0);
    host_mem = runStreamsBlockingKernel2StreamsOptimal(host_mem, num_elements);
    printHostMemory(host_mem, num_elements);

    return 0;
}